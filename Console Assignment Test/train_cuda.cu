#include "hip/hip_runtime.h"
// File: train_cuda.cu
#ifdef __HIPCC__
#include "train_cuda.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void conv_layer_gpu(float* input, float* output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < IMAGE_PIXELS) {
        output[idx] = input[idx] * 0.5f; // dummy operation
    }
}

void train_cuda(const std::vector<Image>& dataset) {
    std::cout << "Training using GPU with CUDA...\n";
    float* d_input, * d_output;
    hipMalloc(&d_input, IMAGE_PIXELS * sizeof(float));
    hipMalloc(&d_output, IMAGE_PIXELS * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int epoch = 0; epoch < 5; ++epoch) {
        for (const auto& img : dataset) {
            hipMemcpy(d_input, img.pixels.data(), IMAGE_PIXELS * sizeof(float), hipMemcpyHostToDevice);
            conv_layer_gpu << <(IMAGE_PIXELS + 255) / 256, 256 >> > (d_input, d_output);
            hipDeviceSynchronize();
        }
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA GPU training time: " << milliseconds / 1000.0f << " seconds\n";

    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
#endif
